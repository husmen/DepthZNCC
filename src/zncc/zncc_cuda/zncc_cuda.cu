#include "hip/hip_runtime.h"
#include "zncc_cuda.cuh"

// Define kernel for calculating mean
__device__ double calculateMean(int x, int y, int d, int width, int height, int halfWinSize, const unsigned char* img)
{
    int yy_0 = max(d, y - halfWinSize);
    int yy_1 = min(height, y + halfWinSize);
    int xx_0 = max(d, x - halfWinSize);
    int xx_1 = min(width - d, x + halfWinSize);

    double sum = 0.0;
    for (int yy = yy_0; yy < yy_1; yy++)
    {
        for (int xx = xx_0; xx < xx_1; xx++)
        {
            sum += img[yy * width + xx - d];
        }
    }

    int count = (xx_1 - xx_0) * (yy_1 - yy_0);
    return sum / (double)count;
}

// Define kernel for calculating ZNCC
__device__ double calculateZncc(int x, int y, int d, double mean1, double mean2, int width, int height, int halfWinSize, const unsigned char* img1, const unsigned char* img2)
{
    int yy_0 = max(d, y - halfWinSize);
    int yy_1 = min(height - d, y + halfWinSize);
    int xx_0 = max(d, x - halfWinSize);
    int xx_1 = min(width - d, x + halfWinSize);

    double num = 0.0;
    double denom1 = 0.0;
    double denom2 = 0.0;

    for (int yy = yy_0; yy < yy_1; yy++)
    {
        for (int xx = xx_0; xx < xx_1; xx++)
        {
            double val1 = img1[yy * width + xx] - mean1;
            double val2 = img2[yy * width + xx - d] - mean2;
            num += val1 * val2;
            denom1 += val1 * val1;
            denom2 += val2 * val2;
        }
    }

    double denom = sqrt(denom1 * denom2);
    double result = denom == 0.0 ? 0.0 : num / denom;
    return result;
}

// Kernel for ZNCC disparity calculation
__global__ void zncc_kernel(const unsigned char* leftImg,
                        const unsigned char* rightImg,
                        unsigned char* disparityImg,
                        double* meanVals,
                        double* znccVals,
                        int width, int height, int winSize, int maxDisp)
{
    // Get global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx % width;
    int y = idx / width;

    double maxZncc = -1.0;
    int bestDisp = 0;

    double meanVals_0 = calculateMean(x, y, 0, width, height, winSize / 2, leftImg);

    for (int d = 0; d < maxDisp; d++)
    {
        meanVals[d] = calculateMean(x, y, d, width, height, winSize / 2, rightImg);
    }

    for (int d = 0; d < maxDisp; d++)
    {
        znccVals[d] = calculateZncc(x, y, d, meanVals_0, meanVals[d], width, height, winSize / 2, leftImg, rightImg);
    }

    for (int d = 0; d < maxDisp; d++)
    {
        if (znccVals[d] > maxZncc)
        {
            maxZncc = znccVals[d];
            bestDisp = d;
        }
    }

    disparityImg[idx] = (unsigned char)bestDisp;
}

void zncc_cuda(unsigned char* dispMap, const unsigned char* leftImg, const unsigned char* rightImg, int width, int height, int winSize, int maxDisp)
{
    // Allocate device memory
    unsigned char* d_leftImg;
    unsigned char* d_rightImg;
    unsigned char* d_disparityImg;
    double* d_meanVals;
    double* d_znccVals;

    hipMalloc((void**)&d_leftImg, width * height * sizeof(unsigned char));
    hipMalloc((void**)&d_rightImg, width * height * sizeof(unsigned char));
    hipMalloc((void**)&d_disparityImg, width * height * sizeof(unsigned char));
    hipMalloc((void**)&d_meanVals, maxDisp * sizeof(double));
    hipMalloc((void**)&d_znccVals, maxDisp * sizeof(double));

    // Copy input data from host to device
    hipMemcpy(d_leftImg, leftImg, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_rightImg, rightImg, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    dim3 blockSize(256);  // Adjust the block size as needed
    dim3 gridSize((width * height + blockSize.x - 1) / blockSize.x);
    zncc_kernel<<<gridSize, blockSize>>>(d_leftImg, d_rightImg, d_disparityImg, d_meanVals, d_znccVals, width, height, winSize, maxDisp);

    // Copy output data from device to host
    hipMemcpy(dispMap, d_disparityImg, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_leftImg);
    hipFree(d_rightImg);
    hipFree(d_disparityImg);
    hipFree(d_meanVals);
    hipFree(d_znccVals);
}
